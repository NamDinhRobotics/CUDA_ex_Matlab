#include "hip/hip_runtime.h"
//
// File: mulMatrix.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 15-Dec-2020 11:16:10
//

// Include Files
#include "mulMatrix.h"
#include "mulMatrix_data.h"
#include "mulMatrix_initialize.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "hip/hip_math_constants.h"

// Function Declarations
static __global__ void mulMatrix_kernel1(const double A[1000000], const double
  B[1000000], double b_A[1000000], double mul[1000000]);
static __global__ void mulMatrix_kernel2(double mul[1000000]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double A[1000000]
//                const double B[1000000]
//                double b_A[1000000]
//                double mul[1000000]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void mulMatrix_kernel1(const double
  A[1000000], const double B[1000000], double b_A[1000000], double mul[1000000])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 1000000) {
    // UNTITLED Summary of this function goes here
    //    Detailed explanation goes here
    mul[i] = B[i];
    b_A[i] = A[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double mul[1000000]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void mulMatrix_kernel2(double mul
  [1000000])
{
  int i;
  i = static_cast<int>(mwGetGlobalThreadIndex());
  if (i < 1000000) {
    mul[i] = HIP_NAN;
  }
}

//
// Add kernelfun pragma to trigger kernel creation
// Arguments    : const double A[1000000]
//                const double B[1000000]
//                double mul[1000000]
// Return Type  : void
//
void mulMatrix(const double A[1000000], const double B[1000000], double mul
               [1000000])
{
  double (*b_gpu_A)[1000000];
  double (*gpu_A)[1000000];
  double (*gpu_B)[1000000];
  double (*gpu_mul)[1000000];
  int (*gpu_IPIV)[1000];
  int info_t;
  int *gpu_info_t;
  if (!isInitialized_mulMatrix) {
    mulMatrix_initialize();
  }

  hipMalloc(&gpu_mul, 8000000UL);
  hipMalloc(&gpu_info_t, 4UL);
  hipMalloc(&gpu_IPIV, 4000UL);
  hipMalloc(&gpu_A, 8000000UL);
  hipMalloc(&gpu_B, 8000000UL);
  hipMalloc(&b_gpu_A, 8000000UL);

  // UNTITLED Summary of this function goes here
  //    Detailed explanation goes here
  hipMemcpy(b_gpu_A, (void *)&A[0], 8000000UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B, (void *)&B[0], 8000000UL, hipMemcpyHostToDevice);
  mulMatrix_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*b_gpu_A,
    *gpu_B, *gpu_A, *gpu_mul);
  hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), 1000, 1000, (double *)
    &(*gpu_A)[0], 1000, getCuSolverWorkspaceReq());
  setCuSolverWorkspaceTypeSize(8);
  cusolverInitWorkspace();
  hipsolverDnDgetrf(getCuSolverGlobalHandle(), 1000, 1000, (double *)&(*gpu_A)[0],
                   1000, (double *)getCuSolverWorkspaceBuff(), &(*gpu_IPIV)[0],
                   gpu_info_t);
  hipMemcpy(&info_t, gpu_info_t, 4UL, hipMemcpyDeviceToHost);
  if (info_t < 0) {
    mulMatrix_kernel2<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_mul);
  } else {
    hipsolverDnDgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N, 1000, 1000, (double
      *)&(*gpu_A)[0], 1000, &(*gpu_IPIV)[0], (double *)&(*gpu_mul)[0], 1000,
                     gpu_info_t);
  }

  cusolverDestroyWorkspace();
  hipMemcpy(&mul[0], gpu_mul, 8000000UL, hipMemcpyDeviceToHost);
  hipFree(*b_gpu_A);
  hipFree(*gpu_B);
  hipFree(*gpu_A);
  hipFree(*gpu_IPIV);
  hipFree(gpu_info_t);
  hipFree(*gpu_mul);
}

//
// File trailer for mulMatrix.cu
//
// [EOF]
//
